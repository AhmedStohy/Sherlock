// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void
vectorAdd(const int *__restrict a, const int *__restrict b, int *__restrict c, int N)
{
	// Calculate global thread ID
	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	// Boundary check
	if (thread_id < N)
		c[thread_id] = a[thread_id] + b[thread_id];
}

// Check vector add result
void
verify_result(const int *a, const int *b, const int *c, int N)
{
	for (int i = 0; i < N; i++)
		assert(a[i] + b[i] == c[i]);
}

int
main(int argc, char const *argv[])
{
	// Array size of 2^16 (65536 elements)
	constexpr int N = 1 << 16;
	constexpr size_t bytes = sizeof(int) * N;

	// Vectors for holding the host-side (CPU-side) data
	int a_h[N], b_h[N], c_h[N];
	// Initialize random numbers in each array
	for (int i = 0; i < N; i++)
	{
		a_h[i] = rand() % 100;
		b_h[i] = rand() % 100;
	}

	// Allocate memory on the device
	int *a_d, *b_d, *c_d;
	hipMalloc((void **)&a_d, bytes);
	hipMalloc((void **)&b_d, bytes);
	hipMalloc((void **)&c_d, bytes);

	// Copy data from the host to the device (CPU -> GPU)
	hipMemcpy(a_d, a_h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, bytes, hipMemcpyHostToDevice);

	// Threads per CTA (1024)
	int NUM_THREADS = 1 << 10;

	// CTAs per Grid
	// We need to launch at LEAST as many threads as we have elements
	// This equation pads an extra CTA to the grid if N cannot evenly be divided
	// by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	// Launch the kernel on the GPU
	// Kernel calls are asynchronous (the CPU program continues execution after
	// call, but no necessarily before the kernel finishes)
	vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(a_d, b_d, c_d, N);
	std::cout << "COMPLETED  ADDING SUCCESSFULLY !" << std::endl;

	// Copy sum vector from device to host
	// cudaMemcpy is a synchronous operation, and waits for the prior kernel
	// launch to complete (both go to the default stream in this case).
	// Therefore, this cudaMemcpy acts as both a memcpy and synchronization
	// barrier.
	hipMemcpy(c_h, c_d, bytes, hipMemcpyDeviceToHost);

	// Check result for errors
	verify_result(a_h, b_h, c_h, N);

	// Free memory on device
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	std::cout << "CODE COMPLETED SUCCESSFULLY !" << std::endl;

	return 0;
}